#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "radix.h"
#include "efficient.h"

# define blockSize 128

namespace StreamCompaction {
    namespace Radix {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        // Compute inverse bits given dev_idata, dev_b, and dev_e.
        // dev_e is inverse of dev_b.
        __global__ void kernComputeInverseBits(int n, int bit, int* idata, int* b, int* e) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);

            if (index >= n) {
                return;
            }

            // Get bit for dev_b via masking.
            int mask = 1 << bit;
            b[index] = (idata[index] & mask) ? 1 : 0;
            // e is inverse of b.
            e[index] = 1 - b[index];
        }

        // Compute each t using current index, f, and totalFalse value.
        __global__ void kernComputeT(int n, int* f, int* t, int totalFalse) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);

            if (index >= n) {
                return;
            }

            t[index] = index - f[index] + totalFalse;
        }

        // Scatter based on address d.
        __global__ void kernScatterRadix(int n, int* d, int* b, int * t, int* f) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);

            if (index >= n) {
                return;
            }

            d[index] = b[index] ? t[index] : f[index];
        }

        /**
         * Implementation of parallel radix sort on GPU.
         */
        void radixSort(int n, int* odata, const int* idata) {

            // Create buffers.
            int* dev_idata;
            int* dev_odata;
            int* dev_b;
            int* dev_e;
            int* dev_f;
            int* dev_t;
            int* dev_d;

            int totalFalse;

            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc failed to create buffer.");

            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc failed to create buffer.");

            hipMalloc((void**)&dev_b, n * sizeof(int));
            checkCUDAError("hipMalloc failed to create buffer.");

            hipMalloc((void**)&dev_e, n * sizeof(int));
            checkCUDAError("hipMalloc failed to create buffer.");

            hipMalloc((void**)&dev_f, n * sizeof(int));
            checkCUDAError("hipMalloc failed to create buffer.");

            hipMalloc((void**)&dev_t, n * sizeof(int));
            checkCUDAError("hipMalloc failed to create buffer.");

            hipMalloc((void**)&dev_d, n * sizeof(int));
            checkCUDAError("hipMalloc failed to create buffer.");

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            timer().startGpuTimer();
            // For loop through int size.
            for (int i = 0; i < sizeof(int); i++) {
                // Step 1: Fill both b and e arrays from idata using kernComputeInverseBits.
                kernComputeInverseBits << < fullBlocksPerGrid, blockSize >> > (n, i, dev_idata, dev_b, dev_e);
                hipDeviceSynchronize();

                // Step 2: Exclusive scan e to obtain f (call Efficient scan from efficient.cu).
                StreamCompaction::Efficient::scan(n, dev_e, dev_f);

                // Step 3: Compute total false.
                
                int lastE = 0;
                int lastF = 0;

                hipMemcpy(&lastE, &dev_e[n - 1], sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(&lastF, &dev_f[n - 1], sizeof(int), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();

                totalFalse = lastE + lastF;

                // Step 4: Use total false to compute t.
                kernComputeT << < fullBlocksPerGrid, blockSize >> > (n, dev_f, dev_t, totalFalse);
                hipDeviceSynchronize();

                // Step 5: Scatter based on address d.
                kernScatterRadix << < fullBlocksPerGrid, blockSize >> > (n, dev_d, dev_b, dev_t, dev_f);
                hipDeviceSynchronize();

                // Swap data points for input of next iteration.
                int* temp = dev_idata;
                dev_idata = dev_odata;
                dev_odata = temp;
            }
            timer().endGpuTimer();

            hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);

            // Free buffers.
            hipFree(dev_idata);
            hipFree(dev_odata);
            hipFree(dev_b);
            hipFree(dev_e);
            hipFree(dev_f);
            hipFree(dev_t);
            hipFree(dev_d);
        }
    }
}
